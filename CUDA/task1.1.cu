/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include <hip/hip_runtime.h>
#include "stdio.h"

#define N 10

__host__ void add(int *a, int *b, int *c)
{
    int id = 0;
    while (id < N)
    {
        c[id] = a[id] + b[id];
        id += 1;
    }
}

__global__ void add_p(int *a, int *b, int *c)
{
    printf("(%d %d)\n", threadIdx.x, blockIdx.x);
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    c[id] = a[id] + b[id];
}


int main( void ) 
{
    /*
     CUDA task1.1
     block/thread parallelize vector sum
     */

    int a[N], b[N], c[N];

    // fill the arrays 'a' and 'b' on the CPU
    for (int i = 0; i < N; i++)
    {
        a[i] = -i;
        b[i] = i * i;
    }

    //device memory allocation
    int *dev_a, *dev_b, *dev_c;
    hipMalloc(&dev_a, sizeof(int) * N);
    hipMalloc(&dev_b, sizeof(int) * N);
    hipMalloc(&dev_c, sizeof(int) * N);

    hipMemcpy(dev_a, a, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, sizeof(int) * N, hipMemcpyHostToDevice);


    // display the host results
    add(a, b, c);

    printf("host:\n");
    for (int i = 0; i < N; i++)
    {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    for (int i = 0; i < N; i++)
    {
        c[i] = 0;
    }

    printf("\n");

    // display the device (threads) results
    printf("device (threads):\n");
    add_p<<<1, N>>>(dev_a, dev_b, dev_c);
    hipMemcpy(c, dev_c, sizeof(int) * N, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++)
    {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    for (int i = 0; i < N; i++)
    {
        c[i] = 0;
        // dev_c[i] = 0;
    }

    printf("\n");

    // display the device (blocks) results
    printf("device (blocks):\n");
    add_p<<<N, 1>>>(dev_a, dev_b, dev_c);
    hipMemcpy(c, dev_c, sizeof(int) * N, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++)
    {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }


    //freeing device memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}